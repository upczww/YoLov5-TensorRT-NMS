#include "hip/hip_runtime.h"
#include <assert.h>
#include "yololayer.h"
#include "utils.h"

using namespace Yolo;

namespace nvinfer1
{
    YoloLayerPlugin::YoloLayerPlugin(int classCount, int netWidth, int netHeight, int maxOut, const std::vector<Yolo::YoloKernel> &vYoloKernel)
    {
        mClassCount = classCount;
        mYoloV5NetWidth = netWidth;
        mYoloV5NetHeight = netHeight;
        mMaxOutObject = maxOut;
        mYoloKernel = vYoloKernel;
        mKernelCount = vYoloKernel.size();

        CUDA_CHECK(hipHostMalloc(&mAnchor, mKernelCount * sizeof(void *)));
        size_t AnchorLen = sizeof(float) * CHECK_COUNT * 2;
        for (int ii = 0; ii < mKernelCount; ii++)
        {
            CUDA_CHECK(hipMalloc(&mAnchor[ii], AnchorLen));
            const auto &yolo = mYoloKernel[ii];
            CUDA_CHECK(hipMemcpy(mAnchor[ii], yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
        }
    }
    YoloLayerPlugin::~YoloLayerPlugin()
    {
        for (int ii = 0; ii < mKernelCount; ii++)
        {
            CUDA_CHECK(hipFree(mAnchor[ii]));
        }
        CUDA_CHECK(hipHostFree(mAnchor));
    }

    // create the plugin at runtime from a byte stream
    YoloLayerPlugin::YoloLayerPlugin(const void *data, size_t length)
    {
        using namespace Tn;
        const char *d = reinterpret_cast<const char *>(data), *a = d;
        read(d, mClassCount);
        read(d, mThreadCount);
        read(d, mKernelCount);
        read(d, mYoloV5NetWidth);
        read(d, mYoloV5NetHeight);
        read(d, mMaxOutObject);
        mYoloKernel.resize(mKernelCount);
        auto kernelSize = mKernelCount * sizeof(YoloKernel);
        memcpy(mYoloKernel.data(), d, kernelSize);
        d += kernelSize;
        CUDA_CHECK(hipHostMalloc(&mAnchor, mKernelCount * sizeof(void *)));
        size_t AnchorLen = sizeof(float) * CHECK_COUNT * 2;
        for (int ii = 0; ii < mKernelCount; ii++)
        {
            CUDA_CHECK(hipMalloc(&mAnchor[ii], AnchorLen));
            const auto &yolo = mYoloKernel[ii];
            CUDA_CHECK(hipMemcpy(mAnchor[ii], yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
        }
        assert(d == a + length);
    }

    void YoloLayerPlugin::serialize(void *buffer) const
    {
        using namespace Tn;
        char *d = static_cast<char *>(buffer), *a = d;
        write(d, mClassCount);
        write(d, mThreadCount);
        write(d, mKernelCount);
        write(d, mYoloV5NetWidth);
        write(d, mYoloV5NetHeight);
        write(d, mMaxOutObject);
        auto kernelSize = mKernelCount * sizeof(YoloKernel);
        memcpy(d, mYoloKernel.data(), kernelSize);
        d += kernelSize;

        assert(d == a + getSerializationSize());
    }

    size_t YoloLayerPlugin::getSerializationSize() const
    {
        return sizeof(mClassCount) + sizeof(mThreadCount) + sizeof(mKernelCount) + sizeof(Yolo::YoloKernel) * mYoloKernel.size() + sizeof(mYoloV5NetWidth) + sizeof(mYoloV5NetHeight) + sizeof(mMaxOutObject);
    }

    int YoloLayerPlugin::initialize()
    {
        return 0;
    }

    Dims YoloLayerPlugin::getOutputDimensions(int index, const Dims *inputs, int nbInputDims)
    {   assert(index < 2);
        //output the result to channel
        if (index == 0)
        {
            return Dims3(mMaxOutObject, 1, 4);
        }
        return DimsHW(mMaxOutObject, mClassCount);
    }

    // Set plugin namespace
    void YoloLayerPlugin::setPluginNamespace(const char *pluginNamespace)
    {
        mPluginNamespace = pluginNamespace;
    }

    const char *YoloLayerPlugin::getPluginNamespace() const
    {
        return mPluginNamespace;
    }

    // Return the DataType of the plugin output at the requested index
    DataType YoloLayerPlugin::getOutputDataType(int index, const nvinfer1::DataType *inputTypes, int nbInputs) const
    {
        return DataType::kFLOAT;
    }

    // Return true if output tensor is broadcast across a batch.
    bool YoloLayerPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool *inputIsBroadcasted, int nbInputs) const
    {
        return false;
    }

    // Return true if plugin can use input that is broadcast across batch without replication.
    bool YoloLayerPlugin::canBroadcastInputAcrossBatch(int inputIndex) const
    {
        return false;
    }

    void YoloLayerPlugin::configurePlugin(const PluginTensorDesc *in, int nbInput, const PluginTensorDesc *out, int nbOutput)
    {
    }

    // Attach the plugin object to an execution context and grant the plugin the access to some context resource.
    void YoloLayerPlugin::attachToContext(cudnnContext *cudnnContext, cublasContext *cublasContext, IGpuAllocator *gpuAllocator)
    {
    }

    // Detach the plugin object from its execution context.
    void YoloLayerPlugin::detachFromContext() {}

    const char *YoloLayerPlugin::getPluginType() const
    {
        return "YoloLayer_TRT";
    }

    const char *YoloLayerPlugin::getPluginVersion() const
    {
        return "1";
    }

    void YoloLayerPlugin::destroy()
    {
        delete this;
    }

    // Clone the plugin
    IPluginV2IOExt *YoloLayerPlugin::clone() const
    {
        YoloLayerPlugin *p = new YoloLayerPlugin(mClassCount, mYoloV5NetWidth, mYoloV5NetHeight, mMaxOutObject, mYoloKernel);
        p->setPluginNamespace(mPluginNamespace);
        return p;
    }

    __device__ float Logist(float data) { return 1.0f / (1.0f + expf(-data)); };

    __global__ void CalDetection(const float *input, float *bboxData, float *scoreData, int *countData, int noElements,
                                 const int netwidth, const int netheight, int maxoutobject, int yoloWidth, int yoloHeight, const float anchors[CHECK_COUNT * 2], int classes)
    {
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= noElements)
            return;

        int total_grid = yoloWidth * yoloHeight;
        int bnIdx = idx / total_grid;
        idx = idx - total_grid * bnIdx;
        int info_len_i = 5 + classes;                                                    // 85
        const float *curInput = input + bnIdx * (info_len_i * total_grid * CHECK_COUNT); // b*h*w*3*85

        for (int k = 0; k < 3; ++k)
        {
            float box_prob = Logist(curInput[idx + k * info_len_i * total_grid + 4 * total_grid]);
            if (box_prob < IGNORE_THRESH)
                continue;
            int *res_count = countData + bnIdx;
            int count = (int)atomicAdd(res_count, 1);
            if (count >= maxoutobject)
                return;

            float *curBbox = bboxData + bnIdx * maxoutobject * 4 + count * 4;
            float *curScore = scoreData + bnIdx * maxoutobject * classes + count * classes;

            for (int i = 5; i < info_len_i; ++i)
            {
                float p = Logist(curInput[idx + k * info_len_i * total_grid + i * total_grid]);
                curScore[i - 5] = p * box_prob;
            }
            int row = idx / yoloWidth;
            int col = idx % yoloWidth;

            //Location
            // pytorch:
            //  y = x[i].sigmoid()
            //  y[..., 0:2] = (y[..., 0:2] * 2. - 0.5 + self.grid[i].to(x[i].device)) * self.stride[i]  # xy
            //  y[..., 2:4] = (y[..., 2:4] * 2) ** 2 * self.anchor_grid[i]  # wh
            //  X: (sigmoid(tx) + cx)/FeaturemapW *  netwidth
            float cx = (col - 0.5f + 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 0 * total_grid])) * netwidth / yoloWidth;
            float cy = (row - 0.5f + 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 1 * total_grid])) * netheight / yoloHeight;

            // W: (Pw * e^tw) / FeaturemapW * netwidth
            // v5: https://github.com/ultralytics/yolov5/issues/471
            float w = 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 2 * total_grid]);
            w = w * w * anchors[2 * k];
            float h = 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 3 * total_grid]);
            h = h * h * anchors[2 * k + 1];
            // cx,cy,w,h to x1,y1,x2,y2
            curBbox[0] = cx - 0.5 * w;
            curBbox[1] = cy - 0.5 * h;
            curBbox[2] = cx + 0.5 * w;
            curBbox[3] = cy + 0.5 * h;
        }
    }

    void YoloLayerPlugin::forwardGpu(const float *const *inputs, void **outputs, void *workspace, hipStream_t stream, int batchSize)
    {
        float *bboxData = (float *)outputs[0];
        float *scoreData = (float *)outputs[1];
        int *countData = (int *)workspace;

        for (int idx = 0; idx < batchSize; ++idx)
        {
            CUDA_CHECK(hipMemset(countData + idx, 0, sizeof(int)));
        }
        CUDA_CHECK(hipMemset(bboxData, 0, sizeof(float) * mMaxOutObject * 4 * batchSize));
        CUDA_CHECK(hipMemset(scoreData, 0, sizeof(float) * mMaxOutObject * mClassCount * batchSize));

        int numElem = 0;
        for (unsigned int i = 0; i < mYoloKernel.size(); ++i)
        {
            const auto &yolo = mYoloKernel[i];
            numElem = yolo.width * yolo.height * batchSize;
            if (numElem < mThreadCount)
                mThreadCount = numElem;
            CalDetection<<<(yolo.width * yolo.height * batchSize + mThreadCount - 1) / mThreadCount, mThreadCount>>>(inputs[i], bboxData, scoreData, countData, numElem, mYoloV5NetWidth, mYoloV5NetHeight, mMaxOutObject, yolo.width, yolo.height, (float *)mAnchor[i], mClassCount);
        }
    }

    int YoloLayerPlugin::enqueue(int batchSize, const void *const *inputs, void **outputs, void *workspace, hipStream_t stream)
    {
        forwardGpu((const float *const *)inputs, outputs, workspace, stream, batchSize);
        return 0;
    }

    PluginFieldCollection YoloPluginCreator::mFC{};
    std::vector<PluginField> YoloPluginCreator::mPluginAttributes;

    YoloPluginCreator::YoloPluginCreator()
    {
        mPluginAttributes.clear();

        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char *YoloPluginCreator::getPluginName() const
    {
        return "YoloLayer_TRT";
    }

    const char *YoloPluginCreator::getPluginVersion() const
    {
        return "1";
    }

    const PluginFieldCollection *YoloPluginCreator::getFieldNames()
    {
        return &mFC;
    }

    IPluginV2IOExt *YoloPluginCreator::createPlugin(const char *name, const PluginFieldCollection *fc)
    {
        int class_count = 80;
        int input_w = 416;
        int input_h = 416;
        int max_output_object_count = 1000;
        std::vector<Yolo::YoloKernel> yolo_kernels(3);

        const PluginField *fields = fc->fields;
        for (int i = 0; i < fc->nbFields; i++)
        {
            if (strcmp(fields[i].name, "netdata") == 0)
            {
                assert(fields[i].type == PluginFieldType::kFLOAT32);
                int *tmp = (int *)(fields[i].data);
                class_count = tmp[0];
                input_w = tmp[1];
                input_h = tmp[2];
                max_output_object_count = tmp[3];
            }
            else if (strstr(fields[i].name, "yolodata") != NULL)
            {
                assert(fields[i].type == PluginFieldType::kFLOAT32);
                int *tmp = (int *)(fields[i].data);
                YoloKernel kernel;
                kernel.width = tmp[0];
                kernel.height = tmp[1];
                for (int j = 0; j < fields[i].length - 2; j++)
                {
                    kernel.anchors[j] = tmp[j + 2];
                }
                yolo_kernels[2 - (fields[i].name[8] - '1')] = kernel;
            }
        }
        YoloLayerPlugin *obj = new YoloLayerPlugin(class_count, input_w, input_h, max_output_object_count, yolo_kernels);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2IOExt *YoloPluginCreator::deserializePlugin(const char *name, const void *serialData, size_t serialLength)
    {
        // This object will be deleted when the network is destroyed, which will
        // call YoloLayerPlugin::destroy()
        YoloLayerPlugin *obj = new YoloLayerPlugin(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }
} // namespace nvinfer1
